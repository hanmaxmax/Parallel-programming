#include "hip/hip_runtime.h"
﻿//# include <sys/time.h>
#include <windows.h>
#include <iostream>
#include<stdio.h>
#include<stdlib.h>
#include <iomanip>
#include <sstream>
#include <fstream>
#include "hip/hip_runtime.h"
#include ""



using namespace std;


unsigned int Act[8399*264] = { 0 };
unsigned int Pas[8399*264] = { 0 };

const int Num = 263;
const int pasNum = 4535;
const int lieNum = 8399;


//
//unsigned int Act[23045*722] = { 0 };
//unsigned int Pas[23045*722] = { 0 };
//
//const int Num = 721;
//const int pasNum = 14325;
//const int lieNum = 23045;


//
//unsigned int Act[37960*1188] = { 0 };
//unsigned int Pas[37960*1188] = { 0 };
//
//const int Num = 1187;
//const int pasNum = 14921;
//const int lieNum = 37960;
//



//unsigned int Act[43577*1363] = { 0 };
//unsigned int Pas[54274*1363] = { 0 };
//
//const int Num = 1362;
//const int pasNum = 54274;
//const int lieNum = 43577;
//

//消元子初始化
void init_A()
{
    //每个消元子第一个为1位所在的位置，就是它所在二维数组的行号
    //例如：消元子（561，...）由Act[561][]存放
    unsigned int a;
    ifstream infile("act.txt");
    char fin[10000] = { 0 };
    int index;
    //从文件中提取行
    while (infile.getline(fin, sizeof(fin)))
    {
        std::stringstream line(fin);
        int biaoji = 0;

        //从行中提取单个的数字
        while (line >> a)
        {
            if (biaoji == 0)
            {
                //取每行第一个数字为行标
                index = a;
                biaoji = 1;
            }
            int k = a % 32;
            int j = a / 32;

            int temp = 1 << k;
            Act[index * (Num + 1) + Num - 1 - j] += temp;
            Act[index * (Num + 1) + Num] = 1;//设置该位置记录消元子该行是否为空，为空则是0，否则为1
        }
    }
}

//被消元行初始化
void init_P()
{
    //直接按照磁盘文件的顺序存，在磁盘文件是第几行，在数组就是第几行
    unsigned int a;
    ifstream infile("pas.txt");
    char fin[10000] = { 0 };
    int index = 0;
    //从文件中提取行
    while (infile.getline(fin, sizeof(fin)))
    {
        std::stringstream line(fin);
        int biaoji = 0;

        //从行中提取单个的数字
        while (line >> a)
        {
            if (biaoji == 0)
            {
                //用Pas[ ][263]存放被消元行每行第一个数字，用于之后的消元操作
                Pas[index * (Num + 1) + Num] = a;
                biaoji = 1;
            }

            int k = a % 32;
            int j = a / 32;

            int temp = 1 << k;
            Pas[index * (Num + 1) + Num - 1 - j] += temp;
        }
        index++;
    }
}



__global__ void work(int g_Num, int g_pasNum, int g_lieNum, int* g_Act, int* g_Pas)
{
    int g_index = blockIdx.x * blockDim.x + threadIdx.x;
    int gridStride = gridDim.x * blockDim.x;

    for (int i = g_lieNum - 1; i - 8 >= -1; i -= 8)
    {
        //每轮处理8个消元子，范围：首项在 i-7 到 i

        for (int j = g_index; j < g_pasNum; j+=gridStride)
        {
            //看被消元行有没有首项在此范围内的
            while (g_Pas[j * (g_Num + 1) + g_Num] <= i && g_Pas[j * (Num + 1) + g_Num] >= i - 7)
            {
                int index = g_Pas[j * (Num + 1) + g_Num];

                if (g_Act[index * (Num + 1) + g_Num] == 1)//消元子不为空
                {
                    //Pas[j][]和Act[（Pas[j][x]）][]做异或
                    //*******************SIMD优化部分***********************
                    //********
                    for (int k = 0; k < g_Num; k ++)
                    {
                        g_Pas[j * (Num + 1) + k] = g_Pas[j * (Num + 1) + k] ^ g_Act[index * (Num + 1) + k];
                    }
                    //*******
                    //********************SIMD优化部分***********************


                    //更新Pas[j][18]存的首项值
                    //做完异或之后继续找这个数的首项，存到Pas[j][18]，若还在范围里会继续while循环
                    //找异或之后Pas[j][ ]的首项
                    int num = 0, S_num = 0;
                    for (num = 0; num < g_Num; num++)
                    {
                        if (g_Pas[j * (Num + 1) + num] != 0)
                        {
                            unsigned int temp = g_Pas[j * (Num + 1) + num];
                            while (temp != 0)
                            {
                                temp = temp >> 1;
                                S_num++;
                            }
                            S_num += num * 32;
                            break;
                        }
                    }
                    g_Pas[j * (Num + 1) + g_Num] = S_num - 1;
                }
                else//消元子为空
                {
                    break;
                }
            }
        }
    }

    for (int i = g_lieNum % 8 - 1; i >= 0; i--)
    {
        //每轮处理1个消元子，范围：首项等于i

        for (int j = g_index; j < g_pasNum; j+=gridStride)
        {
            //看53个被消元行有没有首项等于i的
            while (g_Pas[j * (Num + 1) + g_Num] == i)
            {
                if (g_Act[i * (Num + 1) + g_Num] == 1)//消元子不为空
                {
                    //Pas[j][]和Act[i][]做异或
                    //*******************SIMD优化部分***********************
                    //********
                    for (int k = 0; k < g_Num; k ++)
                    {
                        g_Pas[j * (Num + 1) + k] = g_Pas[j * (Num + 1) + k] ^ g_Act[i * (Num + 1) + k];
                    }
                    //*******
                    //********************SIMD优化部分***********************

                    //更新Pas[j][18]存的首项值
                    //做完异或之后继续找这个数的首项，存到Pas[j][18]，若还在范围里会继续while循环
                    //找异或之后Pas[j][ ]的首项
                    int num = 0, S_num = 0;
                    for (num = 0; num < g_Num; num++)
                    {
                        if (g_Pas[j * (Num + 1) + num] != 0)
                        {
                            unsigned int temp = g_Pas[j * (Num + 1) + num];
                            while (temp != 0)
                            {
                                temp = temp >> 1;
                                S_num++;
                            }
                            S_num += num * 32;
                            break;
                        }
                    }
                    g_Pas[j * (Num + 1) + g_Num] = S_num - 1;

                }
                else//消元子为空
                {
                    break;
                }
            }
        }
    }

}


//__global__ void upgrade(int g_Num, int g_pasNum, int g_lieNum, int* g_Act, int* g_Pas)
//{
//    
//    printf("%d\n", g_Pas[2 * (g_Num + 1) + g_Num]);
//    g_Pas[2 * (g_Num + 1) + g_Num] = 100;
//
//}


int main()
{
    hipError_t ret;

    init_A();
    init_P();
    

   /* for (int i = 0; i < lieNum; i++)
    {
        for (int j = 0; j < Num + 1; j++)
        {
            cout << Pas[i * (Num + 1) + j] << " ";
        }
        cout << endl;
    }*/



    //cout << Pas[2 * (Num + 1) + Num] << " ";


    int* g_Act, *g_Pas;
    
    ret=hipMalloc(&g_Act, lieNum * (Num + 1) * sizeof(int));
    ret=hipMalloc(&g_Pas, lieNum * (Num + 1) * sizeof(int));
    if (ret != hipSuccess) {
        printf("hipMalloc gpudata failed!\n");
    }


    

    size_t threads_per_block = 256;
    size_t number_of_blocks = 32;


    hipEvent_t start, stop;//计时器
    float etime = 0.0;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);//开始计时

    bool sign;
    do
    {
        ret = hipMemcpy(g_Act, Act, sizeof(int) * lieNum * (Num + 1), hipMemcpyHostToDevice);
        ret = hipMemcpy(g_Pas, Pas, sizeof(int) * lieNum * (Num + 1), hipMemcpyHostToDevice);
        if (ret != hipSuccess) {
            printf("hipMemcpyHostToDevice failed!\n");
        }
        

        //不升格地处理被消元行------------
        work <<< number_of_blocks, threads_per_block >>> (Num, pasNum, lieNum, g_Act, g_Pas);
        //work << < 1024, 100 >> > (Num, pasNum, lieNum, g_Act, g_Pas);

        hipDeviceSynchronize();
        //不升格地处理被消元行------------


        ret = hipMemcpy(Act, g_Act, sizeof(int) * lieNum * (Num + 1), hipMemcpyDeviceToHost);
        ret = hipMemcpy(Pas, g_Pas, sizeof(int) * lieNum * (Num + 1), hipMemcpyDeviceToHost);
        if (ret != hipSuccess) {
            printf("hipMemcpyDeviceToHost failed!\n");
        }

        //升格消元子，然后判断是否结束
        sign = false;
        for (int i = 0; i < pasNum; i++)
        {
            //找到第i个被消元行的首项
            int temp = Pas[i* (Num + 1) + Num];
            if (temp == -1)
            {
                //说明他已经被升格为消元子了
                continue;
            }
            //看这个首项对应的消元子是不是为空，若为空，则补齐
            if (Act[temp * (Num + 1) + Num] == 0)
            {
                //补齐消元子
                for (int k = 0; k < Num; k++)
                    Act[temp * (Num + 1) + k] = Pas[i * (Num + 1) + k];
                //将被消元行升格
                Pas[i * (Num + 1) + Num] = -1;
                //标志bool设为true，说明此轮还需继续
                sign = true;
            }
        }
    } while (sign == true);
    

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);//停止计时
    hipEventElapsedTime(&etime, start, stop);
    printf("GPU_LU:%f ms\n", etime);




}




